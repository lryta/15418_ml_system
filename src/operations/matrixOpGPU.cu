#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>
#include <cstdio>

#include "operations/matrixOpGPU.h"

namespace TinyML{

namespace matrix{

#define pos(i, j, col_num) ((col_num) * (i) + (j))

__global__ void gemmKernel(float *alpha, float *beta, float *omega, float *gamma,
    size_t gamma_row, size_t dim_num, size_t gamma_col,
    bool t_alpha = false,  bool t_beta = false,
    float a = 1, float b = 1) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;

  float value = 0;
  for (int k = 0; k < dim_num; ++k) {
    value += alpha[t_alpha?pos(k, i, gamma_row):pos(i, k, dim_num)]
      * beta[t_beta?pos(j, k, dim_num):pos(k, j, gamma_col)];
  }
  gamma[pos(i, j, gamma_col)] = value * a + ((omega!=NULL)?(omega[j] * b):(0));
}

/* General matrix multiplication
 *  gamma = a * alpha * beta + omega * b
 *
 * Specification:
 *  - alpha(gamma_row, dim_num), row & col exchange on t_alpha
 *  - beta(dim_num, gamma_col), row & col exchange on t_alpha
 *  - omega(gamma_col)
 *  - gamma(gamma_row, gamma_col)
 *
 * Description:
 *  - Omega could be NULL
 */
void gemmGPU(float *alpha, float *beta, float *omega, float *gamma,
    size_t gamma_row, size_t dim_num, size_t gamma_col,
    bool t_alpha,  bool t_beta,
    float a, float b) {
  // 256 threads per block is a healthy number
  int threadsPerBlockx = 16;
  int threadsPerBlocky = 2;
  if (gamma_col % 4 == 0)
    threadsPerBlocky = 4;
  if (gamma_col % 16 == 0)
    threadsPerBlocky = 16;
  dim3 threadsPerBlock(threadsPerBlockx, threadsPerBlocky);

  // 256 block
  dim3 blocks(gamma_row/threadsPerBlockx, gamma_col/threadsPerBlocky);

  gemmKernel<<<blocks, threadsPerBlock>>>(alpha, beta, omega, gamma, gamma_row, dim_num, gamma_col, t_alpha, t_beta, a, b);

  hipDeviceSynchronize();
}

}
}
