#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include "tensor.h"
#include "cstdio"

namespace TinyML {
using std::vector;

float* tensor::getGPUData() {
  if (data_gpu_ == NULL)
    hipMalloc(&data_gpu_, sizeof(float) * shape_.getTotal());
  return data_gpu_;
}

float* tensor::getGPUGrad() {
  if (grad_gpu_ == NULL)
    hipMalloc((void**)&grad_gpu_, sizeof(float) * shape_.getTotal());
  return grad_gpu_;
}

void tensor::SyncDataCPUToGPU() {
  if (data_gpu_ == NULL)
    hipMalloc(&data_gpu_, sizeof(float) * shape_.getTotal());
  hipMemcpy(data_gpu_, data_, sizeof(float) * shape_.getTotal(), hipMemcpyHostToDevice);
}

void tensor::SyncDataGPUToCPU() {
  assert(data_gpu_ != NULL && data_ != NULL);
  hipMemcpy(data_, data_gpu_, sizeof(float) * shape_.getTotal(), hipMemcpyDeviceToHost);
}

void tensor::SyncGradCPUToGPU() {
  assert(grad_ != NULL);
  if (grad_gpu_ == NULL)
    hipMalloc(&grad_gpu_, sizeof(float) * shape_.getTotal());
  hipMemcpy(grad_gpu_, grad_, sizeof(float) * shape_.getTotal(), hipMemcpyHostToDevice);
}

void tensor::SyncGradGPUToCPU() {
  assert(grad_gpu_ != NULL);
  getGrad();
  // make sure grad is allocated;
  hipMemcpy(grad_, grad_gpu_, sizeof(float) * shape_.getTotal(), hipMemcpyDeviceToHost);
}

}
